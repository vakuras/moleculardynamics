#include "hip/hip_runtime.h"
///
/// CUDA Device Functions
/// 
/// Molecular Dynamics Simulation on GPU
///
/// Written by Vadim Kuras. 2009-2010.
///

#include "..\..\common\Constants.h"

texture<int, 2, hipReadModeElementType> ljTexRef; //lennrad jones texture reference
texture<int, 2, hipReadModeElementType> mbTexRef; //many body texture reference

//more functions -> merged together to one cuda module
#include "Getters.cuh"
#include "arvdwfc.cuh"
#include "BondOrderFuncs.cuh"
#include "LennardJones.cuh"
#include "Information.cuh"

///
/// Acceleration Calculation Kernel
///
extern "C" __global__ void calculateAccelerations(float4 * posArray, float3 * forceArray, float3 * aaccArray, int NumberOfParticles)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id>=NumberOfParticles) //more thread than particles
		return;

	//read global memory
	float3 f = forceArray[id];
	float3 a;
	float4 r = posArray[id];

	float mass = getMass((int)r.w); //get mass for particle

	//calculate accelerations
	a.x = f.x/mass;
	a.y = f.y/mass;
	a.z = f.z/mass;

	//write memory
	aaccArray[id] = a;
}

///
/// Kernel to handle the predict function.
///
extern "C" __global__ void predict(float4 * posArray, float3 * velocityArray, float3 * aaccArray, float3 * baccArray, float3 * caccArray, float dt, int nop)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id>=nop) //more thread than particles
		return;

	//read memory
	float4 r = posArray[id];
	float3 v = velocityArray[id];
	float3 a = aaccArray[id];
	float3 b = baccArray[id];
	float3 c = caccArray[id];

	//predict...
	float c1 = dt;
    float c2 = __fmul_rn(c1,dt)/2.0f;
    float c3 = __fmul_rn(c2,dt)/3.0f;
    float c4 = __fmul_rn(c3,dt)/4.0f;

    r.x += __fmul_rn(c1,v.x) + __fmul_rn(c2,a.x) + __fmul_rn(c3,b.x) + __fmul_rn(c4,c.x);
	r.y += __fmul_rn(c1,v.y) + __fmul_rn(c2,a.y) + __fmul_rn(c3,b.y) + __fmul_rn(c4,c.y);
	r.z += __fmul_rn(c1,v.z) + __fmul_rn(c2,a.z) + __fmul_rn(c3,b.z) + __fmul_rn(c4,c.z);
	v.x += __fmul_rn(c1,a.x) + __fmul_rn(c2,b.x) + __fmul_rn(c3,c.x);
	v.y += __fmul_rn(c1,a.y) + __fmul_rn(c2,b.y) + __fmul_rn(c3,c.y);    
	v.z += __fmul_rn(c1,a.z) + __fmul_rn(c2,b.z) + __fmul_rn(c3,c.z);    
	a.x += __fmul_rn(c1,b.x) + __fmul_rn(c2,c.x);
	a.y += __fmul_rn(c1,b.y) + __fmul_rn(c2,c.y); 
	a.z += __fmul_rn(c1,b.z) + __fmul_rn(c2,c.z);
	b.x += __fmul_rn(c1,c.x);
	b.y += __fmul_rn(c1,c.y);
	b.z += __fmul_rn(c1,c.z);

	//write memory
	posArray[id] = r;
	velocityArray[id] = v;
	aaccArray[id] = a;
	baccArray[id] = b;
	caccArray[id] = c;
}

///
/// Kernel to handle the correct function.
///
extern "C" __global__ void correct(float4 * posArray, float3 * velocityArray, float3 * forceArray, float3 * aaccArray, float3 * baccArray, float3 * caccArray, float dt, int nop, float nPos, bool * flag, float energyLoss)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id>=nop) //more thread than particles
		return;

	//read memory
	float4 r = posArray[id];
	float3 v = velocityArray[id];
	float3 f = forceArray[id];
	float3 a = aaccArray[id];
	float3 b = baccArray[id];
	float3 c = caccArray[id];

	float mass = getMass(r.w); //get mass

	float c1 = dt;
    float c2 = __fmul_rn(c1,dt)/2.0f;
    float c3 = __fmul_rn(c2,dt)/3.0f;
    float c4 = __fmul_rn(c3,dt)/4.0f;

	float cr = __fmul_rn(GEAR1,c2);
	float cv = __fmul_rn(GEAR2,c2)/c1;
	float cb = __fmul_rn(GEAR3,c2)/c3;
	float cc = __fmul_rn(GEAR4,c2)/c4;

	float axi = f.x/mass;
  	float ayi = f.y/mass;
  	float azi = f.z/mass;

	float corrx = axi - a.x;
	float corry = ayi - a.y;
	float corrz = azi - a.z;

	r.x += __fmul_rn(cr,corrx);
	r.y += __fmul_rn(cr,corry);
	r.z += __fmul_rn(cr,corrz);
	v.x += __fmul_rn(cv,corrx);
	v.y += __fmul_rn(cv,corry);
	v.z += __fmul_rn(cv,corrz);
	a.x = axi;
	a.y = ayi;
	a.z = azi;
	b.x += __fmul_rn(cb,corrx);
	b.y += __fmul_rn(cb,corry);
	b.z += __fmul_rn(cb,corrz);
	c.x += __fmul_rn(cc,corrx);
	c.y += __fmul_rn(cc,corry);
	c.z += __fmul_rn(cc,corrz);

	//out of the box check
	if (r.x > nPos && r.y > 0)
	{
		*flag = true;
		v.x = -v.x*energyLoss;
		r.x = nPos;
	}

	if (r.x > nPos && r.y < 0)
	{
		*flag = true;
		r.x = -nPos;
	}

	if (r.x < -nPos)
	{
		*flag = true;
		v.x = -v.x;
		r.x = -nPos;
	}

	if (fabs(r.y) > nPos)
	{
		*flag = true;
		v.y = -v.y;
		r.y = __fmul_rn(nPos, (r.y/fabs(r.y)));
	}

	if (fabs(r.z) > nPos)
	{
		*flag = true;
		v.z = -v.z;
		r.z = __fmul_rn(nPos, (r.z/fabs(r.z)));
	}

	//write memory
	posArray[id] = r;
	velocityArray[id] = v;
	aaccArray[id] = a;
	baccArray[id] = b;
	caccArray[id] = c;
}